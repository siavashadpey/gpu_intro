
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
__global__ void add(int *a, int *b, int *c) {
*c = *a + *b;
}
int main() {
int a, b, c;
// host copies of variables a, b & c
int *d_a, *d_b, *d_c;
// device copies of variables a, b & c
int size = sizeof(int);
// Allocate space for device copies of a, b, c
hipMalloc((void **)&d_a, size);
hipMalloc((void **)&d_b, size);
hipMalloc((void **)&d_c, size);
// Setup input values  
c = 0;
a = 3;
b = 5;
// Copy inputs to device
hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
// Launch add() kernel on GPU
add<<<1,1>>>(d_a, d_b, d_c);
// Copy result back to host
hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
  if(err!=hipSuccess) {
      printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
  }
printf("result is %d. It should be %d. \n",c, a+b);
// Cleanup
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}
